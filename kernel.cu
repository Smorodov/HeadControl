#include "hip/hip_runtime.h"
#include <iostream>

#include <vector>
#include <stdio.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""
#include <opencv2/gpu/devmem2d.hpp> 
#include <stdio.h>

using namespace std;
using namespace cv;
using namespace cv::gpu;

#define PI 3.1415926535897932f

// these exist on the GPU side
texture<float4,2>  texImage;

#define BLCKDIM_X 8
#define BLCKDIM_Y 8

//-----------------------------------------------------------
//
//-----------------------------------------------------------
__global__ void ToTexture(DevMem2Df src,float* frame_dev) 
{        
	// ���������� �������� �������
	const int ix = blockIdx.x * blockDim.x + threadIdx.x;
	const int iy = blockIdx.y * blockDim.y + threadIdx.y;
	// ������ � ������ �������� �����������
	const int imageW=src.cols;
	const int imageH=src.rows;
	// ��������� �� ����������� �� ��
	if (ix < imageW && iy < imageH)
	{
	frame_dev[iy*4*imageW+ix*4]=src.ptr(iy)[ix*3];
	frame_dev[iy*4*imageW+ix*4+1]=src.ptr(iy)[ix*3+1];
	frame_dev[iy*4*imageW+ix*4+2]=src.ptr(iy)[ix*3+2];
	frame_dev[iy*4*imageW+ix*4+3]=0;
	}
}
//-----------------------------------------------------------
//
//-----------------------------------------------------------
float* MatToTexture(const DevMem2Df src,texture<float4,2> &texSrc,int BLOCKDIM_X=8,int BLOCKDIM_Y=8)
{
	float* frame_dev=0;
	// ��������� ������������ �����
	dim3 block(BLOCKDIM_X,BLOCKDIM_Y);
	dim3 grid(ceil((float)src.step/(float)block.x), ceil(((float)src.rows)/(float)block.y));
	hipSetDevice(0);
	//-----------------------------------------------------------
	// 
	//-----------------------------------------------------------
	int size=src.cols*src.rows;
	hipMalloc((void**)&frame_dev, size * sizeof(float4));
	hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
    hipBindTexture2D( 0, texSrc, frame_dev, desc, src.cols, src.rows, src.cols*sizeof(float4) );
	ToTexture<<<grid, block>>>(src,frame_dev);
	hipDeviceSynchronize();		
	return frame_dev;
}

//-----------------------------------------------------------
// ������ ��� ����� �������
//-----------------------------------------------------------
__global__ void kernel(DevMem2Df src, PtrStepf dst) 
{        
	// ���������� �������� �������
	const int ix = blockIdx.x * blockDim.x + threadIdx.x;
	const int iy = blockIdx.y * blockDim.y + threadIdx.y;
	// ������ � ������ �������� �����������
	const int imageW=src.cols;
	const int imageH=src.rows;
	// ��������� �� ����������� �� ��
	if (ix < imageW && iy < imageH)
	{
        float4 clr = tex2D(texImage, ix, iy);
	
	// <<-----------------���� ������ ���

		dst.ptr(iy)[ix*3]=clr.x;
		dst.ptr(iy)[ix*3+1]=clr.y;
		dst.ptr(iy)[ix*3+2]=clr.z;
	}
}

//-----------------------------------------------------------
//
//-----------------------------------------------------------
hipError_t VoterKernelHelper(const DevMem2Df src, PtrStepf dst)
{
	// ������ Succsess :)
	hipError_t cudaStatus=hipSuccess;
	// ---------------------
	dim3 block(BLCKDIM_X,BLCKDIM_Y);
	// ---------------------
	dim3 grid(ceil((float)src.step/(float)block.x), ceil(((float)src.rows)/(float)block.y));
	// �� ������ ���������� ������ �� �������
	float *frame_dev=0;
	frame_dev=MatToTexture(src,texImage);
	// ����� ������� �������
	kernel<<<grid, block>>>(src, dst);
	// ������������� ������� �� ������� (�.�. ���� ��� ���)
	hipDeviceSynchronize();
	// ������� ��������
	hipUnbindTexture( texImage );
	// ���������� ������ �� �������
	hipFree(frame_dev);
	return cudaStatus;
}
